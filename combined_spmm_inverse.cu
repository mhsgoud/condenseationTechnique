// combined_spmm_inverse.cu
// Compile: nvcc -O3 -std=c++17 -arch=native -o combined_spmm_inverse combined_spmm_inverse.cu -lcublas -lcudart -lcusparse
//
// Usage:
//   ./combined_spmm_inverse sparse.txt indx_b.txt idx_std_loc.txt B1 B2
//
// B1 = number of batches in indx_b.txt (these will be inverted)
// B2 = number of batches in idx_std_loc.txt (these will be used as sparse A batches)

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#define CHECK_CUDA(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define CHECK_CUBLAS(call) do { \
    hipblasStatus_t s = (call); \
    if (s != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error %s:%d: %d\n", __FILE__, __LINE__, (int)s); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define CHECK_CUSPARSE(call) do { \
    hipsparseStatus_t s = (call); \
    if (s != HIPSPARSE_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSPARSE error %s:%d: %d\n", __FILE__, __LINE__, (int)s); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Kernel from your SpMM+accumulate code to compute D = C * A^T
__global__
void csr_row_accumulate_columns_kernel(
    int m,                    // rows in A, rows in C (and number of rows in D)
    int n,                    // cols in A, cols in C
    const int* __restrict__ A_rowptr,
    const int* __restrict__ A_cols,
    const float* __restrict__ A_vals,
    const float* __restrict__ C, // dense m x n (column-major, ld = m)
    float* __restrict__ D       // dense m x m (column-major, ld = m); D(:,j) computed here
) {
    // compute column j as a 2D-grid flatten
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if (j >= m) return;

    int tid = threadIdx.x;
    int tcount = blockDim.x;

    int start = A_rowptr[j];
    int end   = A_rowptr[j + 1];

    for (int row = tid; row < m; row += tcount) {
        float acc = 0.0f;
        for (int p = start; p < end; ++p) {
            int colk = A_cols[p];       // column index k
            float aval = A_vals[p];     // A(j,k)
            // C is m x n in column-major, column colk starts at C + colk * m
            acc += aval * C[row + (size_t)colk * m];
        }
        D[row + (size_t)j * m] = acc;
    }
}

// ---------------- IO and helpers ----------------
void read_sparse_matrix(const std::string &fname,int &m,int &n,int &nnz,
                        std::vector<int> &row_ptr,std::vector<int> &cols,std::vector<float> &vals)
{
    std::ifstream fin(fname);
    if(!fin){ std::cerr<<"Cannot open "<<fname<<"\n"; exit(1);}
    fin>>m>>n>>nnz;
    row_ptr.resize(m+1);
    for(int i=0;i<=m;i++) fin>>row_ptr[i];
    cols.resize(nnz);
    for(int i=0;i<nnz;i++) fin>>cols[i];
    vals.resize(nnz);
    for(int i=0;i<nnz;i++) fin>>vals[i];
}

void read_batches_file(const std::string &fname,int B,
                  std::vector<std::vector<int>> &row_batches,
                  std::vector<std::vector<int>> &col_batches)
{
    row_batches.clear();
    col_batches.clear();
    std::ifstream fin(fname);
    if(!fin){ std::cerr<<"Cannot open "<<fname<<"\n"; exit(1);}
    std::string line;
    int b=0;
    while(std::getline(fin,line)){
        if(line.empty()) continue;
        size_t sep=line.find('|');
        if(sep==std::string::npos){ std::cerr<<"Missing | in line in "<<fname<<"\n"; exit(1);}
        std::string left=line.substr(0,sep);
        std::string right=line.substr(sep+1);
        std::stringstream lss(left),rss(right);
        std::vector<int> rows,cols;
        int v;
        while(lss>>v) rows.push_back(v);
        while(rss>>v) cols.push_back(v);
        row_batches.push_back(rows);
        col_batches.push_back(cols);
        b++;
    }
    if(b!=B) std::cerr<<"Warning: expected "<<B<<" batches in "<<fname<<", found "<<b<<"\n";
}

// Extract dense submatrix but **column-major** layout (so it's ready for cuBLAS/cuSPARSE)
std::vector<float> extract_dense_submatrix_colmajor(
    const std::vector<int> &row_ptr,
    const std::vector<int> &cols,
    const std::vector<float> &vals,
    const std::vector<int> &row_indices,
    const std::vector<int> &col_indices)
{
    int nr = (int)row_indices.size();
    int nc = (int)col_indices.size();
    if(nr!=nc){
        std::cerr<<"Submatrix not square ("<<nr<<"x"<<nc<<")\n";
        exit(1);
    }

    int max_col = nc? *std::max_element(col_indices.begin(), col_indices.end()) : -1;
    std::vector<int> col_map(max_col+1, -1);
    for(int j=0;j<nc;j++){
        col_map[col_indices[j]] = j;
    }

    std::vector<float> dense((size_t)nr*nc, 0.0f);
    // store column-major: element (r,c) -> dense[c*nr + r]
    for(int r=0;r<nr;r++){
        int i = row_indices[r];
        for(int jj = row_ptr[i]; jj < row_ptr[i+1]; ++jj){
            int c = cols[jj];
            if(c>=0 && c <= max_col){
                int cc = col_map[c];
                if(cc!=-1) dense[(size_t)cc*nr + r] = vals[jj];
            }
        }
    }
    return dense;
}

void write_matrix_to_file(const std::string &fname, const float* data, int rows, int cols) {
    std::ofstream fout(fname);
    if(!fout) {
        std::cerr << "Cannot open file " << fname << " for writing\n";
        return;
    }
    for(int r = 0; r < rows; r++){
        for(int c = 0; c < cols; c++){
            fout << data[c*(size_t)rows + r] << " ";
        }
        fout << "\n";
    }
    fout.close();
}

template<typename T>
void write_vector_to_file(const std::string &fname, const std::vector<T> &vec) {
    std::ofstream fout(fname);
    if(!fout) {
        std::cerr << "Cannot open file " << fname << " for writing\n";
        return;
    }
    for(auto &v : vec) fout << v << " ";
    fout << "\n";
    fout.close();
}

int main(int argc, char** argv){
    if(argc < 6){
        std::cerr<<"Usage: "<<argv[0]<<" sparse.txt indx_b.txt idx_std_loc.txt B1 B2\n";
        return 1;
    }
    std::string sparse_file = argv[1];
    std::string indx_b_file  = argv[2]; // we will invert these
    std::string idx_std_file = argv[3]; // sparse A used for SpMM
    int B1 = std::stoi(argv[4]); // number of batches in indx_b (to invert)
    int B2 = std::stoi(argv[5]); // number of batches in idx_std_loc (sparse A batches)

    // Read full sparse matrix (CSR) used for extraction
    int m_global,n_global,nnz_global;
    std::vector<int> h_row_ptr, h_cols;
    std::vector<float> h_vals;
    read_sparse_matrix(sparse_file, m_global, n_global, nnz_global, h_row_ptr, h_cols, h_vals);
    std::cout<<"Read sparse matrix m="<<m_global<<", n="<<n_global<<", nnz="<<nnz_global<<"\n";

    // Read batches: indx_b (to invert) and idx_std_loc (sparse A to use)
    std::vector<std::vector<int>> rows_b1, cols_b1;
    std::vector<std::vector<int>> rows_b2, cols_b2;
    read_batches_file(indx_b_file, B1, rows_b1, cols_b1);
    read_batches_file(idx_std_file, B2, rows_b2, cols_b2);

    // -------------------------
    // Extract dense matrices for B1 (to be inverted). store column-major
    // -------------------------
    std::vector<std::vector<float>> dense_list; dense_list.reserve(B1);
    int nsub = -1;
    for(int b=0;b<B1;b++){
        if(rows_b1[b].size() != cols_b1[b].size()){
            std::cerr<<"Error: indx_b batch "<<b<<" is not square\n"; return 1;
        }
        if(nsub==-1) nsub = (int)rows_b1[b].size();
        else if((int)rows_b1[b].size() != nsub){
            std::cerr<<"Error: indx_b batch "<<b<<" has different size than previous\n"; return 1;
        }
        auto dense = extract_dense_submatrix_colmajor(h_row_ptr, h_cols, h_vals, rows_b1[b], cols_b1[b]);
        dense_list.push_back(std::move(dense));
    }
    if(nsub<=0){ std::cerr<<"No submatrices to invert\n"; return 1; }

    std::cout<<"Extracted "<<dense_list.size()<<" dense submatrices of size "<<nsub<<"x"<<nsub<<"\n";

    // -------------------------
    // Invert dense_list using cuBLAS batched routines
    // We will pack them into pinned host memory in column-major order (already col-major)
    // -------------------------
    int batchSize = (int)dense_list.size();
    size_t matrix_elems = (size_t)nsub * nsub;
    size_t total_elems = matrix_elems * batchSize;
    size_t bytes = total_elems * sizeof(float);

    float* h_A = nullptr;
    CHECK_CUDA(hipHostMalloc((void**)&h_A, bytes, hipHostMallocDefault)); // pinned
    for(int b=0;b<batchSize;b++){
        memcpy(h_A + (size_t)b*matrix_elems, dense_list[b].data(), matrix_elems*sizeof(float));
    }

    // device allocations for inversion
    float* d_Ablock = nullptr;
    float* d_Cblock = nullptr; // will hold inverses (column-major)
    CHECK_CUDA(hipMalloc((void**)&d_Ablock, bytes));
    CHECK_CUDA(hipMalloc((void**)&d_Cblock, bytes));
    CHECK_CUDA(hipMemcpy(d_Ablock, h_A, bytes, hipMemcpyHostToDevice));

    // create pointer arrays for batched cuBLAS (device pointers)
    std::vector<float*> h_Aarray(batchSize), h_Carray(batchSize);
    for(int i=0;i<batchSize;i++){
        h_Aarray[i] = d_Ablock + (size_t)i * matrix_elems;
        h_Carray[i] = d_Cblock + (size_t)i * matrix_elems;
    }
    float** d_Aarray = nullptr;
    float** d_Carray = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_Aarray, batchSize * sizeof(float*)));
    CHECK_CUDA(hipMalloc((void**)&d_Carray, batchSize * sizeof(float*)));
    CHECK_CUDA(hipMemcpy(d_Aarray, h_Aarray.data(), batchSize * sizeof(float*), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Carray, h_Carray.data(), batchSize * sizeof(float*), hipMemcpyHostToDevice));

    // cuBLAS handle
    hipblasHandle_t cublasHandle = nullptr;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    // pivot and info arrays
    int* d_Pivots = nullptr;
    int* d_info = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_Pivots, batchSize * nsub * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_info, batchSize * sizeof(int)));

    // events & run LU + inverse
    hipEvent_t e0, e1; CHECK_CUDA(hipEventCreate(&e0)); CHECK_CUDA(hipEventCreate(&e1));
    float ms=0.f;

    CHECK_CUDA(hipEventRecord(e0));
    CHECK_CUBLAS(hipblasSgetrfBatched(cublasHandle, nsub, d_Aarray, nsub, d_Pivots, d_info, batchSize));
    CHECK_CUDA(hipEventRecord(e1));
    CHECK_CUDA(hipEventSynchronize(e1)); CHECK_CUDA(hipEventElapsedTime(&ms, e0, e1));
    std::cout<<"getrfBatched: "<<ms<<" ms\n";

    CHECK_CUDA(hipEventRecord(e0));
    CHECK_CUBLAS(hipblasSgetriBatched(cublasHandle, nsub, (const float**)d_Aarray, nsub, d_Pivots, d_Carray, nsub, d_info, batchSize));
    CHECK_CUDA(hipEventRecord(e1));
    CHECK_CUDA(hipEventSynchronize(e1)); CHECK_CUDA(hipEventElapsedTime(&ms, e0, e1));
    std::cout<<"getriBatched: "<<ms<<" ms\n";

    // Copy inverses back to host pinned if you want (optional). We'll keep device pointer d_Cblock.
    float* h_C = nullptr;
    CHECK_CUDA(hipHostMalloc((void**)&h_C, bytes, hipHostMallocDefault));
    CHECK_CUDA(hipMemcpy(h_C, d_Cblock, bytes, hipMemcpyDeviceToHost));
    std::cout<<"Inversion complete. First inverse (5x5 snippet, column-major):\n";
    for(int r=0;r<std::min(5,nsub);r++){
        for(int c=0;c<std::min(5,nsub);c++){
            // column-major layout: (r,c) at h_C[c*nsub + r]
            printf("%9.6f ", h_C[c*(size_t)nsub + r]);
        }
        printf("\n");
    }

    for (int b = 0; b < B1; b++)
    {
        std::string fname = "out/inverse_batch_" + std::to_string(b) + ".txt";
        write_matrix_to_file(fname, h_C + (size_t)b * matrix_elems, nsub, nsub);
    }
    std::cout << "Inverted matrices written to files.\n";

    // -------------------------
    // Now process idx_std_loc batches: treat each A (sparse) and use corresponding inverse as B.
    // Requirements: B_num_cols == nsub (square inverse). Also B2 <= B1.
    // -------------------------
    if(B2 > B1){
        std::cerr<<"Error: number of sparse batches (B2="<<B2<<") is greater than inverted batches (B1="<<B1<<").\n";
        return 1;
    }

    // Read idx_std_loc batches and prepare CSR and placeholders
    // We'll build device arrays similar to your earlier SpMM program.
    // For simplicity, read the idx_std_file batches using the same read_batches_file (they're already read into rows_b2,cols_b2).
    // But we need per-batch CSR arrays extracted from the global CSR (like your extractor).
    std::vector<int> A_num_rows(B2), A_num_cols(B2), A_nnz(B2);
    std::vector<std::vector<int>> hA_csrOffsets(B2), hA_columns(B2);
    std::vector<std::vector<float>> hA_values(B2);

    for(int b=0;b<B2;b++){
        int rows = (int)rows_b2[b].size();
        int cols = (int)cols_b2[b].size();
        // Build mapping from global col index to local col index (we assume cols_b2[b] contains column indices)
        // We'll produce CSR for the submatrix that picks rows rows_b2[b] and columns cols_b2[b].
        // This builds CSR by scanning each row and keeping entries whose col is in cols_b2[b].
        int max_col = cols? *std::max_element(cols_b2[b].begin(), cols_b2[b].end()) : -1;
        std::vector<int> col_map(max_col+1, -1);
        for(int j=0;j<cols;j++) col_map[cols_b2[b][j]] = j;
        // Build CSR
        hA_csrOffsets[b].resize(rows+1);
        hA_columns[b].clear();
        hA_values[b].clear();
        int accum = 0;
        for(int ri=0; ri<rows; ++ri){
            int global_row = rows_b2[b][ri];
            hA_csrOffsets[b][ri] = accum;
            for(int jj = h_row_ptr[global_row]; jj < h_row_ptr[global_row+1]; ++jj){
                int gc = h_cols[jj];
                if(gc >= 0 && gc <= max_col && col_map[gc] != -1){
                    hA_columns[b].push_back(col_map[gc]);
                    hA_values[b].push_back(h_vals[jj]);
                    accum++;
                }
            }
        }
        hA_csrOffsets[b][rows] = accum;
        A_num_rows[b] = rows;
        A_num_cols[b] = cols;
        A_nnz[b] = accum;
        std::cout<<"Prepared sparse batch "<<b<<": rows="<<rows<<", cols="<<cols<<", nnz="<<accum<<"\n";
        // size check: cols must equal nsub so A*B is valid where B is nsub x nsub
        if(cols != nsub){
            std::cerr<<"Error: idx_std_loc batch "<<b<<" has cols="<<cols<<" but inverted matrices are "<<nsub<<".\n";
            return 1;
        }
    }

    for (int b = 0; b < B2; b++)
    {
        write_vector_to_file("out/batch_" + std::to_string(b) + "_values.txt", hA_values[b]);
        write_vector_to_file("out/batch_" + std::to_string(b) + "_columns.txt", hA_columns[b]);
        write_vector_to_file("out/batch_" + std::to_string(b) + "_rowptr.txt", hA_csrOffsets[b]);
    }
    std::cout << "CSR batch arrays written to files.\n";

    // cuSPARSE + cuBLAS handle for SpMM stage
    hipsparseHandle_t cusHandle = nullptr;
    CHECK_CUSPARSE(hipsparseCreate(&cusHandle));
    // reuse cublasHandle if needed (we already created it)

    // Device pointers for idx_std_loc batches
    std::vector<int*>   dA_csrOffsets(B2);
    std::vector<int*>   dA_columns(B2);
    std::vector<float*> dA_values(B2);
    std::vector<float*> dB_ptrs(B2); // these will point into d_Cblock offsets (inverses)
    std::vector<float*> dC_ptrs(B2);
    std::vector<float*> dD_ptrs(B2);

    std::vector<hipsparseSpMatDescr_t> matA(B2);
    std::vector<hipsparseDnMatDescr_t> matB(B2);
    std::vector<hipsparseDnMatDescr_t> matC(B2);

    // For outputs C and D we'll allocate fresh device buffers per batch
    for(int b=0;b<B2;b++){
        int rows = A_num_rows[b];
        int cols = A_num_cols[b];
        int nnz = A_nnz[b];

        CHECK_CUDA(hipMalloc(&dA_csrOffsets[b], (rows + 1) * sizeof(int)));
        CHECK_CUDA(hipMalloc(&dA_columns[b], nnz * sizeof(int)));
        CHECK_CUDA(hipMalloc(&dA_values[b], nnz * sizeof(float)));

        // dB_ptrs: point into d_Cblock (inverses). choose inverse index b
        dB_ptrs[b] = d_Cblock + (size_t)b * matrix_elems; // each inverse is nsub*nsub floats, column-major

        // allocate output C (rows x nsub) and D (rows x rows)
        CHECK_CUDA(hipMalloc(&dC_ptrs[b], rows * (size_t)nsub * sizeof(float)));
        CHECK_CUDA(hipMalloc(&dD_ptrs[b], rows * (size_t)rows * sizeof(float)));

        // copy CSR arrays
        CHECK_CUDA(hipMemcpy(dA_csrOffsets[b], hA_csrOffsets[b].data(), (rows+1)*sizeof(int), hipMemcpyHostToDevice));
        if(nnz>0) {
            CHECK_CUDA(hipMemcpy(dA_columns[b], hA_columns[b].data(), nnz*sizeof(int), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(dA_values[b], hA_values[b].data(), nnz*sizeof(float), hipMemcpyHostToDevice));
        }

        // create descriptors
        CHECK_CUSPARSE(hipsparseCreateCsr(&matA[b], rows, cols, nnz,
            dA_csrOffsets[b], dA_columns[b], dA_values[b],
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

        // matB: cols x nsub dense. cols==nsub and we use dB_ptrs[b]
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matB[b], cols, nsub, cols, dB_ptrs[b], HIP_R_32F, HIPSPARSE_ORDER_COL));

        // matC: rows x nsub dense (column-major)
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matC[b], rows, nsub, rows, dC_ptrs[b], HIP_R_32F, HIPSPARSE_ORDER_COL));
        // initialize outputs
        CHECK_CUDA(hipMemset(dC_ptrs[b], 0, rows * (size_t)nsub * sizeof(float)));
        CHECK_CUDA(hipMemset(dD_ptrs[b], 0, rows * (size_t)rows * sizeof(float)));
    }

    // Get buffer size (use first batch as representative)
    float alpha = 1.0f, beta = 0.0f;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(cusHandle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA[0], matB[0], &beta, matC[0], HIP_R_32F,
        HIPSPARSE_SPMM_CSR_ALG2, &bufferSize));
    // allocate buffer large enough for num batches (simple approach)
    void* dBuffer = nullptr;
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize * (size_t)B2));

    // Create streams and run per-batch SpMM + kernel for D = C*A^T
    std::vector<hipStream_t> streams(B2);
    for(int b=0;b<B2;b++){
        CHECK_CUDA(hipStreamCreate(&streams[b]));
        CHECK_CUSPARSE(hipsparseSetStream(cusHandle, streams[b]));
        CHECK_CUBLAS(hipblasSetStream(cublasHandle, streams[b]));

        // SpMM: C = A * B
        void* bufptr = (char*)dBuffer + (size_t)b * bufferSize;
        CHECK_CUSPARSE(hipsparseSpMM(cusHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA[b], matB[b], &beta, matC[b],
            HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, bufptr));

        // Launch kernel to compute D = C * A^T
        int rows = A_num_rows[b];
        int cols = A_num_cols[b];
        const int threads = 256;
        int maxGridX = 65535;
        int gx = std::min(rows, maxGridX);
        int gy = (rows + gx - 1) / gx;
        dim3 grid(gx, gy);
        dim3 block(threads);
        // Note: kernel expects C column-major with leading dim = rows
        csr_row_accumulate_columns_kernel<<<grid, block, 0, streams[b]>>>(
            rows, cols,
            dA_csrOffsets[b],
            dA_columns[b],
            dA_values[b],
            dC_ptrs[b],
            dD_ptrs[b]
        );
        CHECK_CUDA(hipGetLastError());
    }

    // synchronize and copy back results
    for(int b=0;b<B2;b++){
        CHECK_CUDA(hipStreamSynchronize(streams[b]));
        // copy C and D back to host if needed (we'll copy to host vectors)
        int rows = A_num_rows[b];
        // host containers
        std::vector<float> hC(rows * (size_t)nsub);
        std::vector<float> hD(rows * (size_t)rows);
        CHECK_CUDA(hipMemcpy(hC.data(), dC_ptrs[b], rows * (size_t)nsub * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hD.data(), dD_ptrs[b], rows * (size_t)rows * sizeof(float), hipMemcpyDeviceToHost));

        // print small snippets
        std::cout<<"Batch "<<b<<" C (first 4 rows x 4 cols):\n";
        for(int r=0;r<std::min(rows,4);r++){
            for(int c=0;c<std::min(nsub,4);c++){
                // column-major in hC: (r,c) -> hC[c*rows + r]
                std::cout<<hC[c*(size_t)rows + r]<<" ";
            }
            std::cout<<"\n";
        }
        std::cout<<"Batch "<<b<<" D (first 4x4):\n";
        for(int r=0;r<std::min(rows,4);r++){
            for(int c=0;c<std::min(rows,4);c++){
                std::cout<<hD[c*(size_t)rows + r]<<" ";
            }
            std::cout<<"\n";
        }
        // destroy stream
        CHECK_CUDA(hipStreamDestroy(streams[b]));
    }

    // Cleanup
    CHECK_CUDA(hipFree(dBuffer));
    for(int b=0;b<B2;b++){
        hipsparseDestroySpMat(matA[b]);
        hipsparseDestroyDnMat(matB[b]);
        hipsparseDestroyDnMat(matC[b]);
        CHECK_CUDA(hipFree(dA_csrOffsets[b]));
        CHECK_CUDA(hipFree(dA_columns[b]));
        CHECK_CUDA(hipFree(dA_values[b]));
        CHECK_CUDA(hipFree(dC_ptrs[b]));
        CHECK_CUDA(hipFree(dD_ptrs[b]));
    }

    // free inversion allocations
    CHECK_CUDA(hipFree(d_Ablock));
    CHECK_CUDA(hipFree(d_Cblock));
    CHECK_CUDA(hipFree(d_Aarray));
    CHECK_CUDA(hipFree(d_Carray));
    CHECK_CUDA(hipFree(d_Pivots));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipHostFree(h_A));
    CHECK_CUDA(hipHostFree(h_C));

    hipsparseDestroy(cusHandle);
    hipblasDestroy(cublasHandle);

    std::cout<<"Done.\n";
    return 0;
}
